#include "hip/hip_runtime.h"
#include "decode_yolov8.h"

__global__ void decode_yolov8_device_kernel(int batch_size, int  num_class, int topK, float conf_thresh,
	float* src, int srcWidth, int srcHeight, int srcArea,
	float* dst, int dstWidth, int dstHeight, int dstArea)
{
	int dx = blockDim.x * blockIdx.x + threadIdx.x; // "srcArea" dim
	int dy = blockDim.y * blockIdx.y + threadIdx.y; // "batch size" dim
	if (dx >= srcHeight || dy >= batch_size)
	{
		return;
	}
	float* pitem = src + dy * srcArea + dx * srcWidth;
	float* class_confidence = pitem + 4;    // Pr(Class0/Object)
	float confidence = *class_confidence++; // Pr(Class1/Object)
	int label = 0;
	for (int i = 1; i < num_class; ++i, ++class_confidence)
	{
		if (*class_confidence > confidence)
		{
			confidence = *class_confidence;
			label = i;
		}
	}
	if (confidence < conf_thresh)
	{
		return;
	}
	int index = atomicAdd(dst + dy * dstArea, 1);

	if (index >= topK)
	{
		return;
	}
	// xywh -> xyxy
	float cx = *pitem++;
	float cy = *pitem++;
	float width = *pitem++;
	float height = *pitem++;

	float left = cx - width * 0.5f;
	float top = cy - height * 0.5f;
	float right = cx + width * 0.5f;
	float bottom = cy + height * 0.5f;
	float* pout_item = dst + dy * dstArea + 1 + index * dstWidth;
	*pout_item++ = left; // todo
	*pout_item++ = top;
	*pout_item++ = right;
	*pout_item++ = bottom;
	*pout_item++ = confidence;
	*pout_item++ = label;
	*pout_item++ = 1;// 1 = keep, 0 = ignore
}

void yolov8::decodeDevice(utils::InitParameter param, float* src, int srcWidth, int srcHeight, int srcArea, float* dst, int dstWidth, int dstHeight)
{
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size((srcHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int dstArea = 1 + dstWidth * dstHeight;

	decode_yolov8_device_kernel << < grid_size, block_size, 0, nullptr >> > (param.batch_size, param.num_class, param.topK, param.conf_thresh,
		src, srcWidth, srcHeight, srcArea,
		dst, dstWidth, dstHeight, dstArea);
}


__global__ void transpose_device_kernel(int batch_size,
	float* src, int srcWidth, int srcHeight, int srcArea,
	float* dst, int dstWidth, int dstHeight, int dstArea)
{
	int dx = blockDim.x * blockIdx.x + threadIdx.x; // "srcArea" dim
	int dy = blockDim.y * blockIdx.y + threadIdx.y; // "batch size" dim
	if (dx >= dstHeight || dy >= batch_size)
	{
		return;
	}
	float* p_dst_row = dst + dy * dstArea + dx * dstWidth; // row = dx
	float* p_src_col = src + dy * srcArea + dx; // col = dx

	for (int i = 0; i < dstWidth; i++)
	{
		p_dst_row[i] = p_src_col[i * srcWidth];
	}
}

void yolov8::transposeDevice(utils::InitParameter param, 
float* src, int srcWidth, int srcHeight, int srcArea, 
float* dst, int dstWidth, int dstHeight)
{
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size((dstHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int dstArea = dstWidth * dstHeight;

	transpose_device_kernel << < grid_size, block_size, 0, nullptr >> > (param.batch_size,
		src, srcWidth, srcHeight, srcArea,
		dst, dstWidth, dstHeight, dstArea);
}


